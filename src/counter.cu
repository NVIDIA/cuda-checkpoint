/*
 * SPDX-FileCopyrightText: Copyright (c) 2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: LicenseRef-NvidiaProprietary
 *
 * NVIDIA CORPORATION, its affiliates and licensors retain all intellectual
 * property and proprietary rights in and to this material, related
 * documentation and any modifications thereto. Any use, reproduction,
 * disclosure or distribution of this material and related documentation
 * without an express license agreement from NVIDIA CORPORATION or
 * its affiliates is strictly prohibited.
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/types.h>
#include <sys/socket.h>
#include <netinet/in.h>
#include <arpa/inet.h>

#define PORT 10000

__device__ int counter = 100;
__global__ void increment()
{
    counter++;
}

int main(void)
{
    hipFree(0);

    int sock = socket(PF_INET, SOCK_DGRAM, IPPROTO_UDP);
    sockaddr_in addr = {AF_INET, htons(PORT), inet_addr("127.0.0.1")};
    bind(sock, (sockaddr *)&addr, sizeof addr);

    while (true) {
        char buffer[16] = {0};
        sockaddr_in peer = {0};
        socklen_t inetSize = sizeof peer;
        int hCounter = 0;

        recvfrom(sock, buffer, sizeof buffer, 0, (sockaddr *)&peer, &inetSize);

        increment<<<1,1>>>();
        hipMemcpyFromSymbol(&hCounter, HIP_SYMBOL(counter), sizeof counter);

        size_t bytes = sprintf(buffer, "%d\n", hCounter);
        sendto(sock, buffer, bytes, 0, (sockaddr *)&peer, inetSize);
    }
    return 0;
}
